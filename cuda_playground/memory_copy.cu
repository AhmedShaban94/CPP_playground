#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <string>
#include "hip/hip_runtime.h"
#include "" 
#include <stdlib.h>
#include <ctime> 

__global__ void kernel(int* a, int* b, int* c, int count)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x; 
	if (id < count)
	{
		c[id] = a[id] + b[id]; 
	}
}

int main()
{
	srand(time(NULL));
	//init host containers 
	int count = 10;
	int* h_a = new int[count];
	int* h_b = new int[count];
	int* h_c = new int[count]; 

	//init device containers 
	int* d_a;
	int* d_b;
	int* d_c; 

	//filling the host arrays 
	for (int i = 0; i < count; i++)
	{
		h_a[i] = rand() % 1000;
		h_b[i] = rand() % 1000;
	}


	//cuda memory allocation of array (a)
	if (hipMalloc(&d_a, sizeof(int)*count) != hipSuccess)
	{
		delete[] h_a; 
		delete[] h_b;
	}

	//cuda memory allocation of array (b) 
	if (hipMalloc(&d_b, sizeof(int)*count) != hipSuccess)
	{
		delete[] h_a; 
		delete[] h_b; 
		hipFree(d_a); 
	}

	//cuda mmeory allocation of array(c) 
	if (hipMalloc(&d_c, sizeof(int) * count) != hipSuccess)
	{
		delete[] h_a; 
		delete[] h_b; 
		hipFree(d_a); 
		hipFree(d_b); 
 	}
	//cuda memory copy from host to device array (a) 
	if (hipMemcpy(d_a, h_a, sizeof(int) * count, hipMemcpyHostToDevice) != hipSuccess)
	{
		std::cout << "failed to memory copy of array A \n"; 
		delete[] h_a; 
		delete[] h_b;
		hipFree(&d_a); 
		hipFree(&d_b); 
	}

	//cuda memory copy from host to device array (b) 
	if (hipMemcpy(d_b, h_b, sizeof(int) * count, hipMemcpyHostToDevice) != hipSuccess)
	{
		std::cout << "failed to memory copy of array B \n";
		delete[] h_a;
		delete[] h_b;
		hipFree(&d_a);
		hipFree(&d_b);
	}

	//calling the kernel from host to be executed on device 
	kernel << <count / 256 + 1, 256 >> > (d_a, d_b, d_c, count); 

	//cuda memory copy from device to host of array(a)
	if (hipMemcpy(h_a, d_a, sizeof(int) * count, hipMemcpyDeviceToHost) != hipSuccess)
	{
		delete[] h_a; 
		delete[] h_b; 
		hipFree(&d_a); 
		hipFree(&d_b); 
		hipFree(&d_c); 
	}

	//cuda mmeory copy from device to host of array(b)
	if (hipMemcpy(h_b, d_b, sizeof(int) * count, hipMemcpyDeviceToHost) != hipSuccess)
	{
		delete[] h_a; 
		delete[] h_b; 
		hipFree(&d_a); 
		hipFree(&d_b); 
		hipFree(&d_c); 
	}

	//cuda memory copy form device to host of array(c) 
	if (hipMemcpy(h_c, d_c, sizeof(int)*count, hipMemcpyDeviceToHost) != hipSuccess)
	{
		delete[] h_a; 
		delete[] h_b; 
		delete[] h_c; 
		hipFree(d_a); 
		hipFree(d_b); 
		hipFree(d_c); 
	}

	//showing the filled arrays 
	for (int i = 0; i < count; i++)
	{
		std::cout << h_c[i] << " " << "= " << h_a[i] << " + " << h_b[i] << std::endl;
	} 

	delete[] h_a; 
	delete[] h_b; 
	delete[] h_c; 
	hipFree(d_a); 
	hipFree(d_b); 
	hipFree(d_c); 

	return EXIT_SUCCESS; 
}
