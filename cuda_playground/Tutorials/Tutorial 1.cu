#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string>

 
#include <stdlib.h>
#include <ctime> 
#include <math.h>

float* add_cpu(int count, float* arrayA, float* arrayB)
{
	float* arrayC; 
	hipMallocManaged(&arrayC, sizeof(float)*count); 
	for (int i = 0; i < count; i++)
	{
		arrayC[i] = arrayA[i] + arrayB[i]; 
	}
	return arrayC; 
}

__global__ void add_gpu(int count, float* arrayA, float* arrayB)
{ 
	int index = blockIdx.x * blockDim.x + threadIdx.x; 
	int stride = gridDim.x * blockDim.x; 
	//this called grid-stride loop in cuda. WHY we use this shit ? 
	for (int i = index; i < count; i += stride)
		arrayB[i] = arrayA[i] + arrayB[i];
}
int main()
{
	//one million elements 
	int N = 1<<20; 

	//cuda unified memory allocation 
	float* x; 
	float* y;  
	hipMallocManaged(&x, sizeof(float)*N); 
	hipMallocManaged(&y, sizeof(float)*N); 


	//initlization of arrays 
	for (int i = 0; i < N; i++)
	{
		x[i] = 1.0f; 
		y[i] = 2.0f; 
	}

	//run kernel on N elements on CPU
	//float* z = add_cpu(N, x, y); 

	//run kernel on N elements on GPU 
	int blockSize = 256; 
	int numBlock = (N + blockSize - 1) / blockSize; 
	add_gpu <<<numBlock, blockSize>>>(N, x, y); 
	
	//Force CPU to wait until the kernel is done, befor accessing the results for error checking step 
	hipDeviceSynchronize(); 

	//checking error 
	float maxError = 0.0f; 
	for (int i = 0; i < N; i++)
	{
		maxError = fmax(maxError, fabs(y[i] - 3.0f)); //z[i] for cpu code 
	}
	std::cout << "Max Error = " << maxError << std::endl; 

	hipFree(x); 
	hipFree(y);
	//delete[] z; 


	return EXIT_SUCCESS; 
}

