#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string>

 
#include <stdlib.h> 
#include <assert.h>
/*Tutorial 4 from cuda articels series about suing pinned memory directly without using pageable memory.*/
//Note: Don't over_use pinned memory and use it in narrowest limits. 

inline
hipError_t cudaCheck(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
	{
		if (result != cudaSuccess)
		{
			fprintf(stderr, "Cuda runtime error", cudaGetErrorString(result));
			assert(cudaSuccess == result);
		}
	}
#endif 
	return result;
}

void profileCopies(float *h_a, float *h_b, float *d, unsigned int n, const char *desc)
{
	printf("\n%s transfer \n", desc);
	unsigned int bytes = n * sizeof(float);

	//cuda events for timing
	hipEvent_t start, end;
	cudaCheck(hipEventCreate(&start));
	cudaCheck(hipEventCreate(&end));

	cudaCheck(hipEventRecord(start, 0));
	cudaCheck(hipMemcpy(d, h_a, bytes, hipMemcpyHostToDevice));
	cudaCheck(hipEventRecord(end, 0));
	cudaCheck(hipEventSynchronize(end));

	float time;
	cudaCheck(hipEventElapsedTime(&time, start, end));
	printf("Host to Device Bandwidth (Gb/s): %f\n", bytes * 1e-6 / time);

	cudaCheck(hipEventRecord(start, 0));
	cudaCheck(hipMemcpy(h_b, d, bytes, hipMemcpyDeviceToHost));
	cudaCheck(hipEventRecord(end, 0));
	cudaCheck(hipEventSynchronize(end));

	cudaCheck(hipEventElapsedTime(&time, start, end));
	printf("Device to Host Bandwidth (Gb/s): %f\n", bytes * 1e-6 / time);

	//checking transfer 
	for (int i = 0; i < n; i++)
	{
		if (h_a[i] != h_b[i])
		{
			printf("%s transfer failed\n");
			break;
		}
	}
	cudaCheck(hipEventDestroy(start));
	cudaCheck(hipEventDestroy(end));
}

int main()
{
	//number of elements in each array 
	unsigned int numElements = 1024 * 1024 * 4;
	const unsigned int bytes = numElements * sizeof(float);

	//host arrays 
	float *h_aPageable, *h_bPageable;
	float *h_aPinned, *h_bPinned;

	//device array 
	float *d;

	//allocate memory on host and device 
	h_aPageable = new float[bytes];
	h_bPageable = new float[bytes];
	cudaCheck(hipHostMalloc(&h_aPinned, bytes, hipHostMallocDefault));
	cudaCheck(hipHostMalloc(&h_bPinned, bytes, hipHostMallocDefault));
	cudaCheck(hipMalloc(&d, bytes));

	//initilize memory arrays  
	for (int i = 0; i < numElements; i++) h_aPageable[i] = i;
	memcpy(h_aPageable, h_aPageable, bytes);
	memset(h_bPageable, 0, bytes);
	memcpy(h_bPinned, h_bPageable, bytes);

	//getting device properties 
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0); 
	printf("\nDevice Name \"%s\"\n", prop.name);
	printf("\nTransfer size (MB): %d\n", bytes / (1024 * 1024));

	//performing data_transfer to compare speed between paegable and pinned memory 
	profileCopies(h_aPageable, h_bPageable, d, numElements, "Pageable"); 
	profileCopies(h_aPinned, h_bPinned, d, numElements, "Pinned"); 

	//clean up memory 
	delete[] h_aPageable; 
	delete[] h_bPageable; 
	hipHostFree(h_aPinned); 
	hipHostFree(h_bPinned); 
	hipFree(d); 

	return EXIT_SUCCESS; 
}