#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string>
#include "hip/hip_runtime.h"
#include "" 
#include <stdlib.h>
#include <ctime> 
#include <math.h>
//Tutorial 2 "How to Implement Performance Metrics in CUDA C/C++" 
__global__ 
void sexpy(int n,int a, float* x, float* y)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x; 
	int stride = blockDim.x * gridDim.x; 

	for (int i = index; i < n; i += stride)
	{
		y[i] = a*x[i] + y[i]; 
	}
}

int main()
{
	int nElements = 20 * (1 << 20); 

	float *x_h, *y_h, *x_d, *y_d; 
	x_h = new float[nElements]; 
	y_h = new float[nElements]; 

	hipMalloc(&x_d, sizeof(float)*nElements); 
	hipMalloc(&y_d, sizeof(float)*nElements); 

	//fill host array 
	for (int i = 0; i < nElements; i++)
	{
		x_h[i] = 1.0f; 
		y_h[i] = 2.0f; 
	}

	hipEvent_t start, stop; 
	hipEventCreate(&start); 
	hipEventCreate(&stop); 

	//copy data from host to device 
	hipMemcpy(x_d, x_h, nElements * sizeof(float), hipMemcpyHostToDevice); 
	hipMemcpy(y_d, y_h, nElements * sizeof(float), hipMemcpyHostToDevice);

	hipEventRecord(start);
	sexpy << <(nElements + 511) / 512, 256 >> > (nElements, 2.0f, x_d, y_d); 
	hipEventSynchronize(stop);  

	float milliseconds = 0; 
	hipEventElapsedTime(&milliseconds, start, stop); 
	printf("Effective Bandwidth (GB/s): %fn \n", nElements * 4 * 3 / milliseconds / 1e6);
	
	
	hipMemcpy(y_h, y_d, nElements * sizeof(float), hipMemcpyDeviceToHost); 
	for (int i = 0; i < nElements; i++)
	{
		std::cout << float(y_h[i]) << std::endl; 
	}

	delete[] x_h; 
	delete[] y_h; 
	hipFree(x_d); 
	hipFree(y_d); 

	return EXIT_SUCCESS;
}