#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string>
#include "hip/hip_runtime.h"
#include "" 
#include <stdlib.h>
#include <ctime> 
#include <math.h>
//Tutorial 3 "How to Query device properties and handle errors in CUDA C/C++" 

__global__
void saxpy(int n, float a, float* x, float* y)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	int stride = gridDim.x * blockDim.x;
	for (int i = index; i < n; i += stride)
	{
		y[i] = a*x[i] + y[i];
	}
}
int main()
{
	//Querying devices properties 
	int nDevices = 0;
	hipError_t error = hipGetDeviceCount(&nDevices);
	if (error != hipSuccess) printf("%s", hipGetErrorString(error));
	printf("Number of devices = %d\n", nDevices);
	for (int i = 0; i < nDevices; i++)
	{
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Properties of Device number %d :\n", i);
		printf("Device %d name is %s \n", i, prop.name);
		printf("Device %d clock rate = %d KHz\n", i, prop.clockRate);
		printf("Device %d compute capability is %d.%d \n", i, prop.major, prop.minor);
		printf("Device %d memory clockRate = %d KHz\n", i, prop.memoryClockRate);
		printf("Devide %d maximum grid size = %d grid\n", i, prop.maxGridSize);
		printf("Device %d memory bus width = %d bits\n", i, prop.memoryBusWidth);
		printf("Device %d Peak mmeory bandwidth = %f  (GB/s)\n", i, 2.0*prop.memoryClockRate*(prop.memoryBusWidth / 8) / 1.0e6);

		//code can be extended further to print all fields in hipDeviceProp_t struct
		//but deviceQuery in CUDA samples already do this.
		//this code just for demonstration 
	}

	//Handling errors
	int n = 1 << 20;
	float *h_x, *h_y, *d_x, *d_y;
	h_x = new float[n];
	h_y = new float[n];
	hipMalloc(&d_x, n * sizeof(float));
	hipMalloc(&d_y, n * sizeof(float));

	for (int i = 0; i < n; i++)
	{
		h_x[i] = 1.0f;
		h_y[i] = 2.0f;
	}

	hipMemcpy(d_x, h_x, n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, h_y, n * sizeof(float), hipMemcpyHostToDevice);

	saxpy << <(n + 255) / 256, 256 >> > (n, 2.0f, d_x, d_y);

	//Invalid execution configuration parameters, e.g.too many threads per thread block
	//, are reflected in the value of errSync returned by hipGetLastError()
	hipError_t errorSync = hipGetLastError();

	//Asynchronous errors which occur on the device after control is returned to the host,
	//such as out - of - bounds memory accesses, require a synchronization mechanism such as hipDeviceSynchronize()
	//Device Synch is expensiv, so use it with care. Use it in debug mode but avoid it in release mode.
	hipError_t errorAsync = hipDeviceSynchronize();
	if (errorAsync != hipSuccess) printf("%s", hipGetErrorString(errorAsync));
	if (errorSync != hipSuccess) printf("%s", hipGetErrorString(errorSync));

	hipMemcpy(h_y, d_y, n * sizeof(float), hipMemcpyDeviceToHost);

	for (int i = 0; i < n; i++)
	{
		printf("%.1f\n", h_y[i]);
	}

	delete[] h_x;
	delete[] h_y;
	hipFree(d_x);
	hipFree(d_y);

	return EXIT_SUCCESS;
}