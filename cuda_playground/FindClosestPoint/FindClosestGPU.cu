#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <string>
#include "hip/hip_runtime.h"
#include "" 
#include <stdlib.h>
#include <ctime> 

#include"FindClosestCPU.h"

using std::cout;

__global__ void findClosestGPU(float3* points, int* indicies, int count)
{
	if (count <= 1)
		return;

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < count)
	{
		float3 thisPoint = points[idx];
		float closestSoFar = 3.40282e38f;

		for (int i = 0; i < count; i++)
		{
			if (idx == i)
				continue;
			float dist = (thisPoint.x - points[i].x) * (thisPoint.x - points[i].x);
			dist += (thisPoint.y - points[i].y) * (thisPoint.y - points[i].y);
			dist += (thisPoint.z - points[i].z) * (thisPoint.z - points[i].z);
			if (dist < closestSoFar)
			{
				dist = closestSoFar;
				indicies[idx] = i;
			}
		}
	}
}

int main()
{
	float3* d_points; 
	int* d_indexOfClosest; 
	hipMalloc(&d_points, sizeof(float3)); 
	hipMalloc(&d_indexOfClosest, sizeof(int)); 
	//number of points 
	const int count = 10000;

	//arrays of points 
	int* indexOfclosest = new int[count]; 
	float3* points = new float3[count]; 

	//create a list of random points 
	for (int i = 0; i < count; i++)
	{
		points[i].x = (float)(rand() % 10000 - 5000); 
		points[i].y = (float)(rand() % 10000 - 5000); 
		points[i].z = (float)(rand() % 10000 - 5000); 
	}

	hipMemcpy(d_points, points, count, hipMemcpyHostToDevice); 
    hipMemcpy(d_indexOfClosest, indexOfclosest, count, hipMemcpyHostToDevice); 

	//this variable is set to keep track of the fastest time so far 
	long fastestTime = 1000000; 

	//run the algorithm 20 times 
	for (int run = 0; run < 20; run++)
	{
		long startTime = clock(); 
		//findClosestPoint(points, indexOfclosest, count); 
		findClosestGPU << <(count / 32) + 1, 32 >> > (points, indexOfclosest, count); 
		hipMemcpy(indexOfclosest, d_indexOfClosest, sizeof(int)*count, hipMemcpyDeviceToHost); 
		hipMemcpy(points, d_points, sizeof(float3)*count, hipMemcpyDeviceToHost); 
		long endTime = clock(); 
		cout << "Run no. " << run << " took " << endTime - startTime << " millis\n"; 
		//if the run time was fastest that the fastestTime, Update fastestTime. 
		if ((endTime - startTime) < fastestTime)
		{
			fastestTime = endTime - startTime; 
		}
	}
	//print out fastestTime. 
	cout << "Fastest Time is: " << fastestTime << " millis\n";

	//print the final results to screen 
	cout << "final results\n";
	for (int i = 0; i < 10; i++)
	{
		cout << i << ". " << indexOfclosest[i] << std::endl;
	}
	
	//deallocate CPU ram 
	delete[] indexOfclosest;
	delete[] points;

	//deallocate Cuda memory 
	hipFree(d_points); 
	hipFree(d_indexOfClosest); 
	hipDeviceReset(); 

	return EXIT_SUCCESS; 
}


